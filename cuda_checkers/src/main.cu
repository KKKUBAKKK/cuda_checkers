#include <iostream>
#include <thread>
#include <chrono>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>

#include "../include/Game.h"

int main() {
    // Game game = Game::getGameInfo();
    Game game = Game();
    game.run();
    return 0;
}