// #include <iostream>
// #include <thread>
// #include <chrono>
// #include <hip/hip_runtime.h>
// #include <hiprand/hiprand_kernel.h>

#include "../include/Game.h"

int main() {
    // Game game = Game::getGameInfo();
    Game game = Game();
    game.run();
    return 0;
}

// #include <iostream> 
// #include <hip/hip_runtime.h>

// int main() { 
//     int deviceCount = 0; 
//     hipError_t err = hipGetDeviceCount(&deviceCount); 
//     if (err != hipSuccess) { 
//         std::cerr << "hipGetDeviceCount failed: " << hipGetErrorString(err) << std::endl; 
//         return 1; 
//     } 
//     std::cout << "Number of CUDA-capable devices: " << deviceCount << std::endl; 
//     return 0;
// }