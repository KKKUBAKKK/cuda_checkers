#include "hip/hip_runtime.h"
#include "Player.h"
#include <chrono>
#include <iostream>

__global__ void simulate_game_gpu_kernel(Board initial_board, float* results, hiprandState* states, bool is_player_white) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    Move moves[MAX_MOVES];
    Move stack[MAX_MOVES];

    // Each thread gets its own random state
    hiprandState localState = states[tid];

    // Run simulation
    float result = initial_board.simulate_game_gpu(&localState, moves, stack, is_player_white);

    // Store result
    atomicAdd(results, result);

    // Save updated random state
    states[tid] = localState;
}

__global__ void init_curand(hiprandState* state, unsigned long seed) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    hiprand_init(seed, idx, 0, &state[idx]);
}

Player::Player(bool is_white, bool is_cpu, int max_games, int max_iterations, float time_limit_ms) :
                        is_white(is_white), is_cpu(is_cpu), time_limit_ms(time_limit_ms), max_iterations(max_iterations), max_games(max_games) {
    root = new Node(nullptr, true);

    // Initialize random states
    if (!is_cpu) {
        CUDA_CHECK(hipMalloc(&states, max_games * sizeof(hiprandState)));
        int num_blocks = (max_games + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
        init_curand<<<num_blocks, THREADS_PER_BLOCK>>>(states, time(NULL));
        CUDA_CHECK(hipGetLastError());
        CUDA_CHECK(hipDeviceSynchronize());
    } else {
        states = nullptr;
    }
}

Player::Player(Board board, bool is_white, bool is_cpu, 
    int max_games, int max_iterations, 
    float time_limit_ms) :
    is_white(is_white), is_cpu(is_cpu), time_limit_ms(time_limit_ms), 
    max_iterations(max_iterations), max_games(max_games) {
    root = new Node(board, nullptr);

    // Initialize random states
    if (!is_cpu) {
        CUDA_CHECK(hipMalloc(&states, max_games * sizeof(hiprandState)));
        int num_blocks = (max_games + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
        init_curand<<<num_blocks, THREADS_PER_BLOCK>>>(states, time(NULL));
        CUDA_CHECK(hipGetLastError());
        CUDA_CHECK(hipDeviceSynchronize());
    } else {
        states = nullptr;
    }
};

Player::~Player() {
    delete root;
    if (states != nullptr)
        CUDA_CHECK(hipFree(states));
};

int Player::findEqualChild(Board board) {
    if (root == nullptr || root->children.empty()) {
        return -1;
    }

    for (int i = 0; i < root->children.size(); i++) {
        if (root->children[i]->board.is_equal(board)) {
            return i;
        }
    }

    return -1;
}

void Player::move_root(Board startBoard) {

    if (root == nullptr) {
        root = new Node(startBoard, nullptr);
        return;
    }

    if (root->board.is_equal(startBoard)) {
        return;
    }

    int index = findEqualChild(startBoard);
    if (index == -1) {
        delete root;
        root = new Node(startBoard, nullptr);
        return;
    }

    Node *temp = root;
    root = root->children[index];
    root->parent = nullptr;
    temp->children[index] = nullptr;
    delete temp;
}

Node* Player::select() {
    // Select the best child
    // If the child is not fully expanded, return it
    // Otherwise, return the best child of the child
    Node* current = root;
    while (current->is_expanded() && !current->is_end()) {
        Node* best_child = nullptr;
        float best_value = std::numeric_limits<float>::min();

        for (Node* child : current->children) {
            float uct_value = child->get_UCT_value();
            if (uct_value > best_value) {
                best_value = uct_value;
                best_child = child;
            }
        }

        if (best_child == nullptr) {
            break;
        }

        current = best_child;
    }

    return current;
}

Node* Player::expand(Node* node) {
    // Expand the node
    // Create a new node and add it to the tree
    // Return the new node
    Move new_move = node->get_move();
    Board new_board = node->board.apply_move(new_move);
    Node* new_node = new Node(new_board, node);
    node->children.push_back(new_node);
    return new_node;
}

float Player::simulate(Node *node) {
    // Simulate the game
    // Run the game until the end
    // Return the result
    if (is_cpu) {
        return simulate_cpu(node->board);
    }

    return simulate_gpu(node->board);
}

float Player::simulate_cpu(Board board) {
    // Simulate n games on the CPU
    // Run the game until the end
    // Return the result
    Move *moves = new Move[MAX_MOVES];
    Move *stack = new Move[MAX_MOVES];
    std::random_device rd;
    std::mt19937 rng(rd());
    float result = root->board.simulate_n_games_cpu(rng, moves, stack, max_games, time_limit_ms, is_white);
    delete[] moves;
    delete[] stack;
    return result;
}

float Player::simulate_gpu(Board board) {
    // Simulate n games on the GPU
    // Run the game until the end
    // Return the result
    int num_blocks = (max_games + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;

    // Allocate memory for results on the GPU
    float* d_results;
    CUDA_CHECK(hipMalloc(&d_results, sizeof(float)));
    CUDA_CHECK(hipMemset(d_results, 0, sizeof(float)));

    // Launch the kernel
    simulate_game_gpu_kernel<<<num_blocks, THREADS_PER_BLOCK>>>(board, d_results, states, is_white);
    CUDA_CHECK(hipGetLastError());
    CUDA_CHECK(hipDeviceSynchronize());

    // Copy results back to the host
    float h_results;
    CUDA_CHECK(hipMemcpy(&h_results, d_results, sizeof(float), hipMemcpyDeviceToHost));

    // Free allocated memory
    CUDA_CHECK(hipFree(d_results));
    CUDA_CHECK(hipFree(states));

    return h_results;
}

void Player::backpropagate(Node *node, float score) {
    // Backpropagate the results up the tree
    // Update the score and visits of each node
    Node *current = node;
    while (current != nullptr) {
        current->score += score;
        current->visits++;
        current = current->parent;
    }
}

int Player::mcts_loop() {
    auto start_time = std::chrono::high_resolution_clock::now();
    auto time_limit = std::chrono::milliseconds((int) time_limit_ms);

    // Run the MCTS algorithm
    int i = 0;
    for (i = 0; i < max_iterations; i++) {
        // std::cerr << "Iteration " << i << "\n";
        auto current_time = std::chrono::high_resolution_clock::now();
        if (current_time - start_time >= time_limit) {
            break;
        }

        // 1. Selection
        // Find nodes for expansion
        Node *selected = select();
        if (selected->is_end()) {
            float score = selected->white_score();
            assert (score >= 0);
            if (!is_white) score = 1 - score;
            backpropagate(selected, score);
            continue;
        }

        // 2. Expansion
        // For each node, create a new node and add it to the tree
        Node *new_node = expand(selected);

        // 3. Simulation
        // Run simulations on the new nodes
        float score = simulate(new_node);

        // 4. Backpropagation
        // Backpropagate the results up the tree
        backpropagate(new_node, score);
    }

    return i;
}

Node* Player::choose_move() {
    // Find the best child
    Node* best_child = nullptr;
    int best_score = -1;

    for (Node* child : root->children) {
        if (child->score > best_score) {
            best_score = child->score;
            best_child = child;
        }
    }

    return best_child;
}

Board Player::make_move(Board start_board) {
    // Move root to the current board
    move_root(start_board);

    // If root doesn't have any children, return the current board
    if (root->is_end()) {
        return start_board;
    }

    // Run the MCTS algorithm
    mcts_loop();

    // Find the best child
    Node* best_child = choose_move();

    // Set the root to the best child
    move_root(best_child->board);

    // Return the child's board
    return best_child->board;
}